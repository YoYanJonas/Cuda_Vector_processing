
#include "hip/hip_runtime.h"



#include <stdio.h>


// for random initializing 
#include <stdlib.h>
#include <time.h>

// for memset
#include <cstring>


//functions
__global__ void sum_array_cpu_otherthings(int * a, int * b , int * c, long int size)
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    for (int j = 0; j < 10000; j++) {
        if (gid < size) {
            c[gid] = a[gid] + b[gid];
        }
    }
}

/*void compare_arrays(int* a, int* b, long int size)
{
    for (int i = 0; i < size; i++)
    {
        if (a[i] != b[i])
        {
            printf("Arrays are different \n");
            printf("%d - %d | %d \n", i, a[i], b[i]);
            return;
        }
    }
    printf("Arrays are same \n");
}
*/

int main()
{

    //for time measurements
    clock_t start_gpu, end_gpu;
    double CLOCK_PER_SECOND = 1000000.0;
    start_gpu = clock();

    long int size = 1000000;
    int block_size= 128;

    long int NU_BYTES= size * sizeof(long int);

    // host pointers
    int *h_a ,*h_b ,*gpu_results;

    // allocating
    h_a=(int*)malloc(NU_BYTES);
    h_b=(int*)malloc(NU_BYTES);
    // h_c=(int*)malloc(NU_BYTES);
    gpu_results=(int*)malloc(NU_BYTES); 

    //initializing a and b array randomly.
    srand(100);
    for (int i = 0; i < size; i++) {
        h_a[i] = (int)(rand() & 0xFF);
    }
    srand(102);
    for (int i = 0; i < size; i++) {
        h_b[i] = (int)(rand() & 0xFF);
    }


    // sum_array_cpu(h_a,h_b,h_c, size);
    memset(gpu_results, 0, NU_BYTES);

    // device pointer 
    int *d_a,*d_b, *d_c;
    hipMallocManaged((int **)&d_a, NU_BYTES);
    hipMallocManaged((int **)&d_b, NU_BYTES);
    hipMallocManaged((int **)&d_c, NU_BYTES);

    // transfering data
    hipMemcpy(d_a,h_a, NU_BYTES, hipMemcpyHostToDevice);
    hipMemcpy(d_b,h_b, NU_BYTES, hipMemcpyHostToDevice);


    //grid defineing
    dim3  block(block_size);
    dim3 grid((size/block.x)+1); // one more thread to not getting error and guaranteeing having enough threads


    // running and transfering back
    sum_array_cpu_otherthings<<<grid,block>>>(d_a,d_b,d_c,size);
    hipDeviceSynchronize();

    hipMemcpy(gpu_results, d_c, NU_BYTES, hipMemcpyDeviceToHost);

    // array comparing results.
    //compare_arrays(gpu_results, h_c, size);
    
    // freeing memories
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    printf("%d first value of a \n", h_a[0]);
    free(h_a);
    printf("%d first value of b \n", h_b[0]);
    free(h_b);
    printf("%d result of operation(must be a0+b0) \n", gpu_results[0]);
    free(gpu_results);

    end_gpu = clock();
    double time_taken = (end_gpu - start_gpu) / CLOCK_PER_SECOND;
    printf("The taken time for cpu is : %.8f \n", time_taken);

    return 0;

}
